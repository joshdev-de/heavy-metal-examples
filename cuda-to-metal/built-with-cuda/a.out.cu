#include "hip/hip_runtime.h"


/*
 *  Global Switches
 */

#ifndef SAC_DO_CHECK
#define SAC_DO_CHECK                             1
#endif
#ifndef SAC_DO_CHECK_TYPE
#define SAC_DO_CHECK_TYPE                        0
#endif
#ifndef SAC_DO_CHECK_GPU
#define SAC_DO_CHECK_GPU                         1
#endif
#ifndef SAC_DO_CHECK_BOUNDARY
#define SAC_DO_CHECK_BOUNDARY                    0
#endif
#ifndef SAC_DO_CHECK_MALLOC
#define SAC_DO_CHECK_MALLOC                      0
#endif
#ifndef SAC_DO_CHECK_ERRNO
#define SAC_DO_CHECK_ERRNO                       0
#endif
#ifndef SAC_DO_CHECK_HEAP
#define SAC_DO_CHECK_HEAP                        0
#endif
#ifndef SAC_DO_CHECK_DISTMEM
#define SAC_DO_CHECK_DISTMEM                     0
#endif
#ifndef SAC_DO_CHECK_DISTMEMPHM
#define SAC_DO_CHECK_DISTMEMPHM                  0
#endif

#define SAC_DO_PHM                               0
#define SAC_DO_APS                               0
#define SAC_DO_DAO                               0
#define SAC_DO_MSCA                              0
#define SAC_DO_COMPILE_MODULE                    0

#ifndef SAC_DO_PROFILE
#define SAC_DO_PROFILE                           0
#endif
#ifndef SAC_DO_PROFILE_WITH
#define SAC_DO_PROFILE_WITH                      0
#endif
#ifndef SAC_DO_PROFILE_FUN
#define SAC_DO_PROFILE_FUN                       0
#endif
#ifndef SAC_DO_PROFILE_INL
#define SAC_DO_PROFILE_INL                       0
#endif
#ifndef SAC_DO_PROFILE_LIB
#define SAC_DO_PROFILE_LIB                       0
#endif
#ifndef SAC_DO_PROFILE_MEM
#define SAC_DO_PROFILE_MEM                       0
#endif
#ifndef SAC_DO_PROFILE_OPS
#define SAC_DO_PROFILE_OPS                       0
#endif
#ifndef SAC_DO_PROFILE_CUDA
#define SAC_DO_PROFILE_CUDA                      0
#endif
#ifndef SAC_DO_PROFILE_DISTMEM
#define SAC_DO_PROFILE_DISTMEM                   0
#endif

#ifndef SAC_DO_TRACE
#define SAC_DO_TRACE                             1
#endif
#ifndef SAC_DO_TRACE_REF
#define SAC_DO_TRACE_REF                         0
#endif
#ifndef SAC_DO_TRACE_MEM
#define SAC_DO_TRACE_MEM                         0
#endif
#ifndef SAC_DO_TRACE_PRF
#define SAC_DO_TRACE_PRF                         0
#endif
#ifndef SAC_DO_TRACE_FUN
#define SAC_DO_TRACE_FUN                         0
#endif
#ifndef SAC_DO_TRACE_WL
#define SAC_DO_TRACE_WL                          0
#endif
#ifndef SAC_DO_TRACE_AA
#define SAC_DO_TRACE_AA                          0
#endif
#ifndef SAC_DO_TRACE_MT
#define SAC_DO_TRACE_MT                          0
#endif
#ifndef SAC_DO_TRACE_GPU
#define SAC_DO_TRACE_GPU                         1
#endif
#ifndef SAC_DO_TRACE_RTSPEC
#define SAC_DO_TRACE_RTSPEC                      0
#endif
#ifndef SAC_DO_TRACE_DISTMEM
#define SAC_DO_TRACE_DISTMEM                     0
#endif

#ifndef SAC_DO_CACHESIM
#define SAC_DO_CACHESIM                          0
#endif
#ifndef SAC_DO_CACHESIM_ADV
#define SAC_DO_CACHESIM_ADV                      0
#endif
#ifndef SAC_DO_CACHESIM_GLOBAL
#define SAC_DO_CACHESIM_GLOBAL                   1
#endif
#ifndef SAC_DO_CACHESIM_FILE
#define SAC_DO_CACHESIM_FILE                     0
#endif
#ifndef SAC_DO_CACHESIM_PIPE
#define SAC_DO_CACHESIM_PIPE                     0
#endif
#ifndef SAC_DO_CACHESIM_IMDT
#define SAC_DO_CACHESIM_IMDT                     1
#endif

/*
 * Setup for Multi Threaded Data Parallelism
 */
#define SAC_DO_MULTITHREAD                       0
#define SAC_DO_THREADS_STATIC                    1
#define SAC_DO_MT_CREATE_JOIN                    0
#define SAC_DO_MT_PTHREAD                        0
#define SAC_DO_MT_LPEL                           0

/*
 * Setup for OMP Data Parallelism
 */
#define SAC_DO_MT_OMP                            0
#define SAC_DO_OMP_MACROS                        0

/*
 * Setup for MUTC
 */
#define SAC_MUTC_FUNAP_AS_CREATE                 0
#define SAC_MUTC_THREAD_MALLOC                   0
#define SAC_MUTC_DISABLE_THREAD_MEM              0
#define SAC_MUTC_BENCH                           0
#define SAC_MUTC_MACROS                          0
#define SAC_MUTC_RC_INDIRECT                     0
#define SAC_MUTC_SEQ_DATA_PARALLEL               0

/*
 * Setup for GPU Data Parallelism
 */
#define SAC_CUDA_MACROS                          1

/*
 * Setup for Distributed Memory Data Parallelism
 */
#define SAC_DO_DISTMEM                           0
#define SAC_DO_DISTMEM_GASNET                    0
#define SAC_DO_DISTMEM_GPI                       0
#define SAC_DO_DISTMEM_MPI                       0
#define SAC_DO_DISTMEM_ARMCI                     0
#define SAC_DO_DISTMEM_ALLOC_CACHE_OUTSIDE_DSM   0
#define SAC_DO_DISTMEM_PTR_DESC                  0
#define SAC_DO_DISTMEM_PTR_CACHE                 1

/*
 * Setup for Task Parallelism
 */
#define SAC_DO_FP                                0

/*
 * Debugging Support
 */
#ifndef SAC_DEBUG_RC
#define SAC_DEBUG_RC                             0
#endif

#define SAC_DO_CUDA_FORCE_INIT 1


/*
 *  Global Settings
 */

#define SAC_FORCE_DESC_SIZE -1

/*
 *  MUTC Backend Specific Settings
 */
#define SAC_MUTC_RC_PLACES  1
#define SAC_MUTC_FORCE_SPAWN_FLAGS

#define SAC_C_EXTERN           extern "C"



/*
 *  Global Settings
 */

#ifndef NULL
#  ifdef __cplusplus
#    define NULL         0
#  else
#    define NULL         (void*) 0
#  endif
#endif

#define SAC_SET_TMPDIR              "/tmp"
#define SAC_SET_INITIAL_MASTER_HEAPSIZE      1048576
#define SAC_SET_INITIAL_WORKER_HEAPSIZE      65536
#define SAC_SET_INITIAL_UNIFIED_HEAPSIZE     0

#ifndef SAC_SET_RTSPEC_THREADS
#define SAC_SET_RTSPEC_THREADS              1
#endif

#ifndef SAC_SET_MTMODE
#define SAC_SET_MTMODE               0
#endif

#define SAC_SET_CPU_BIND_STRATEGY 0
#define SAC_SET_BARRIER_TYPE               0
#define SAC_SET_SMART_DECISIONS            0
#define SAC_SET_SMART_FILENAME           "default"
#define SAC_SET_SMART_ARCH               "(null)"
#define SAC_SET_SMART_PERIOD               500
#ifndef SAC_SET_THREADS_MAX
#define SAC_SET_THREADS_MAX          128
#endif

#ifndef SAC_SET_THREADS
#define SAC_SET_THREADS              1
#endif

#ifndef SAC_OMP_ACTIVE_LEVEL
#define SAC_OMP_ACTIVE_LEVEL          1
#endif

#ifndef SAC_SET_MASTERCLASS
#define SAC_SET_MASTERCLASS          0
#endif

#define SAC_SET_NUM_SCHEDULERS       0

#define SAC_SET_CACHE_1_SIZE         -1
#define SAC_SET_CACHE_1_LINE         4
#define SAC_SET_CACHE_1_ASSOC        1
#define SAC_SET_CACHE_1_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_1_MSCA_FACTOR  0.00

#define SAC_SET_CACHE_2_SIZE         -1
#define SAC_SET_CACHE_2_LINE         4
#define SAC_SET_CACHE_2_ASSOC        1
#define SAC_SET_CACHE_2_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_2_MSCA_FACTOR  0.00

#define SAC_SET_CACHE_3_SIZE         -1
#define SAC_SET_CACHE_3_LINE         4
#define SAC_SET_CACHE_3_ASSOC        1
#define SAC_SET_CACHE_3_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_3_MSCA_FACTOR  0.00

#define SAC_SET_CACHESIM_HOST        ""
#define SAC_SET_CACHESIM_FILE        "a.out.cs"
#define SAC_SET_CACHESIM_DIR         "/tmp"
#define SAC_SET_MAXFUN               0
#define SAC_SET_MAXFUNAP             1
#define SBLOCKSZ               16
#define LBLOCKSZ               256



/*
 *  Includes
 */


#include "sac.h"


#if SAC_OMP_MACROS

#include "omp.h"

#endif

#if SAC_CUDA_MACROS

#include <stdio.h>


#include <hip/hip_runtime.h>


#include <hip/hip_runtime.h>


#include <algorithm>

#endif

/*
 *  SAC-Program example.sac :
 */


/*
 *  Global Definitions
 */

SAC_PF_DEFINE()
SAC_HM_DEFINE()


/*
 *  prototypes for locals (FUNDEFS)
 */

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf__MAIN__generate, , 1, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf__MAIN__generate, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * CUDA_GLOBALFUN_DECL( SACf__MAIN___cuknl_83_CUDA__id_10000__i, 3, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 1, in, int, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 1, in, int, (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_83_CUDA__id_10000__i(SAC_CUDA_PARAM_out( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), int), int SAC_ND_A_MIRROR_SHAPE((SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SIZE((SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), int), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SIZE((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));
;



/*
 *  function definitions (FUNDEFS)
 */



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN::SACwf__MAIN__main(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()


    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf__MAIN__main, , 1, out, int, SAC_SET_NT_USG( FAG, (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf__MAIN__main,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_17, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN__generate(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN__generate, , 1, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACf__MAIN__generate, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    SAC_ND_DECL_CONST__DATA((SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 42)
    /*
     * ND_DECL( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 10000)
     */
    SAC_ND_DECL__DATA( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 10000;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 10000;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()


    SAC_INIT_LOCAL_MEM()
    SAC_ND_ALLOC_BEGIN((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 10000)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 1), 6, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 10000), 6, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    SAC_ND_ALLOC_BEGIN((SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 6, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 6, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        
{

    SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_84_tmp)
    SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_85_tmp)

SAC_TR_GPU_PRINT("Mapping Gen");
    SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_86_ub)
    SAC_GKCO_OPD_REDEFINE(10000, SAC_gkco_prt_86_ub)

SAC_TR_GPU_PRINT("    Index space (LUSW) = ( [%i, ], [%i, ], [%i, ], [%i, ], )%s", 
0, SAC_gkco_prt_86_ub, 1, 1, "");

SAC_TR_GPU_PRINT("Mapping ShiftLB");
SAC_TR_GPU_PRINT("    Index space (LUSW) = ( [%i, ], [%i, ], [%i, ], [%i, ], )%s", 
0, SAC_gkco_prt_86_ub, 1, 1, "");

SAC_TR_GPU_PRINT("Mapping CompressGrid([1, ])");
SAC_TR_GPU_PRINT("    Index space (LUSW) = ( [%i, ], [%i, ], [%i, ], [%i, ], )%s", 
0, SAC_gkco_prt_86_ub, 1, 1, "");

SAC_TR_GPU_PRINT("Mapping SplitLast(len 32)");
    SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_87_ub)
    SAC_GKCO_HOST_OPD_PAD(SAC_gkco_prt_86_ub, SAC_gkco_prt_86_ub, 32)

    SAC_GKCO_HOST_OPM_SPLIT_LAST(SAC_gkco_prt_86_ub, SAC_gkco_prt_86_ub, SAC_gkco_prt_87_ub, 32)

SAC_TR_GPU_PRINT("    Index space (LUSW) = ( [%i, %i, ], [%i, %i, ], [%i, %i, ], [%i, %i, ], )%s", 
0, 0, SAC_gkco_prt_86_ub, SAC_gkco_prt_87_ub, 1, 1, 1, 1, "");

SAC_TR_GPU_PRINT("Mapping GridBlock (1)");
SAC_TR_GPU_PRINT("    Index space (LUSW) = ( [%i, %i, ], [%i, %i, ], [%i, %i, ], [%i, %i, ], )%s", 
0, 0, SAC_gkco_prt_86_ub, SAC_gkco_prt_87_ub, 1, 1, 1, 1, "");
    SAC_GKCO_HOST_OPM_SET_GRID(2147483647   , 65535   , 65535   , 0, SAC_gkco_prt_86_ub)

    SAC_GKCO_HOST_OPM_SET_BLOCK(1024   , 1024   , 64   , 1024, SAC_gkco_prt_87_ub)

SAC_TR_GPU_PRINT("    Index space (LUSW) = ( [%i, %i, ], [%i, %i, ], [%i, %i, ], [%i, %i, ], )%s", 
0, 0, SAC_gkco_prt_86_ub, SAC_gkco_prt_87_ub, 1, 1, 1, 1, "");


    /*
     * CUDA_GLOBALFUN_AP( SACf__MAIN___cuknl_83_CUDA__id_10000__i, 3, out, int, 1, SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 1, SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
if (block.x * block.y * block.z * grid.x * grid.y * grid.z > 0) {
        SAC_TR_GPU_PRINT ("   kernel name \"SACf__MAIN___cuknl_83_CUDA__id_10000__i\"\n");
SAC_PF_BEGIN_CUDA_KNL ();
SACf__MAIN___cuknl_83_CUDA__id_10000__i<<<grid, block>>>(    SAC_CUDA_ARG_out( SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), 0), SAC_ND_A_MIRROR_SIZE(SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))), SAC_ND_A_MIRROR_DIM(SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), 0), SAC_ND_A_MIRROR_SIZE(SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))), SAC_ND_A_MIRROR_DIM(SAC_SET_NT_USG( FAG, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int));
hipDeviceSynchronize ();
SAC_PF_END_CUDA_KNL ();
SAC_CUDA_GET_LAST_KERNEL_ERROR();
} else {
SAC_TR_GPU_PRINT("Skipping kernel because it has no elements");
SAC_PRAGMA_BITMASK_CHECK_NL
}


    }

    /*
     * ND_REFRESH__MIRROR( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
     */
    SAC_NOOP()

    SAC_ND_FREE((SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACf__MAIN__generate, , 1, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN__main(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    SAC_ND_DECL_CONST__DATA((SACp_emal_82__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 42)
    /*
     * ND_DECL( (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 10000)
     */
    SAC_ND_DECL__DATA( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 10000;
    const int SAC_ND_A_MIRROR_SIZE( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 10000;
    const int SAC_ND_A_MIRROR_DIM( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;


    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf__MAIN__generate, , 1, out, int, SAC_SET_NT_USG( FAG, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf__MAIN__generate,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

    /*
     * ND_REFRESH__MIRROR( (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
     */
    SAC_NOOP()

    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_82__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_82__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_82__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 12, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_82__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    SAC_ND_FREE((SACl_a, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_82__flat_14, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_81__flat_12, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN___cuknl_83_CUDA__id_10000__i(...) [ body ]
 ****************************************************************************/
/*
 * CUDA_GLOBALFUN_DEF_BEGIN( SACf__MAIN___cuknl_83_CUDA__id_10000__i, 3, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 1, in, int, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 1, in, int, (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_83_CUDA__id_10000__i(SAC_CUDA_PARAM_out( (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), int), int SAC_ND_A_MIRROR_SHAPE((SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SIZE((SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), int), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SIZE((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)){

{ 
  /*
   * ND_DECL( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * CUDA_DECL_KERNEL_ARRAY( (SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), int, 1, 1)
   */
  int SAC_ND_A_FIELD( (SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))[1];
  SAC_ND_DECL__DESC( (SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
  const int SAC_ND_A_MIRROR_SHAPE( (SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) = 1;
  const int SAC_ND_A_MIRROR_SIZE( (SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 1;
  const int SAC_ND_A_MIRROR_DIM( (SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 1;


  SAC_INIT_LOCAL_MEM()
  SAC_ND_ALLOC_BEGIN((SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
  /*
   * ND_SET__SHAPE_arr( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
   */
  SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 6, "Assignment with incompatible types found!");
  SAC_NOOP()

  SAC_ND_ALLOC_END((SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_88_tmp)
  SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_89_tmp)
  SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_90_ret_col)

  SAC_GKCO_OPM_RETURN_COL_INIT(SAC_gkco_prt_90_ret_col)




  SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_91_ub)
  SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_92_idx)
  SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_93_ub)
  SAC_GKCO_HOST_OPD_PAD(10000, SAC_gkco_prt_93_ub, 32)

  SAC_GKCO_OPD_DECLARE(SAC_gkco_prt_94_ub)
  SAC_GKCO_HOST_OPM_SPLIT_LAST(SAC_gkco_prt_93_ub, SAC_gkco_prt_94_ub, SAC_gkco_prt_91_ub, 32)



  SAC_GKCO_OPD_REDEFINE(THREADIDX_X, SAC_gkco_prt_92_idx)

  SAC_GKCO_OPD_REDEFINE(BLOCKIDX_X, SAC_ND_READ( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))


  SAC_GKCO_GPUD_OPM_UNSPLIT_LAST(SAC_ND_READ( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_gkco_prt_92_idx, 32)

  SAC_GKCO_GPUD_OPD_UNPAD_BL(10000, SAC_ND_READ( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_gkco_prt_90_ret_col)




  SAC_GKCO_GPUD_OPM_RETURN_IF_COLLECTED(SAC_gkco_prt_90_ret_col)

  SAC_GKCO_GPUD_OPD_DEF_IV((SACp_hzgwl_0, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0, SAC_ND_READ( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))



  /*
   * ND_IDXS2OFFSET_arr( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 10000)
   */
  SAC_ND_WRITE( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = SAC_ND_READ( (SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * CUDA_WL_ASSIGN( (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
   */
  SAC_ND_WRITE_READ_COPY( (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_ND_READ( (SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, );

  SAC_ND_FREE((SACp_wlidx_70__flat_1, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_eat_15, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOOP()

  SAC_CLEANUP_LOCAL_MEM()
}
/*
 * CUDA_GLOBALFUN_DEF_END( SACf__MAIN___cuknl_83_CUDA__id_10000__i, 3, out, int, (SAC_arg_1, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 1, in, int, (SACp_emal_78__iwlmem_77_dev, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), 1, in, int, (SACp_emal_80__flat_4, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
}

/*
 * stubs for SACARGfreeDataUdt and SACARGcopyDataUdt
 */
extern "C" void SACARGfreeDataUdt( int, void *);
extern "C" void *SACARGcopyDataUdt( int, int, void *);
void SACARGfreeDataUdt( int size, void *data) {}
void *SACARGcopyDataUdt( int type, int size, void *data) { return ((void *) 0x0); } 

int main( int __argc, char *__argv[])
{
  SAC_ND_DECL__DATA( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()
  SAC_HWLOC_SETUP();
  SAC_MT_SETUP_INITIAL();
  SAC_PF_SETUP();
  SAC_HM_SETUP();
  SAC_MT_SETUP();
  SAC_CUDA_SETUP();
  SAC_CS_SETUP();
  SAC_COMMANDLINE_SET( __argc, __argv);

  SAC_INVOKE_MAIN_FUN( SACf__MAIN__main, SAC_ND_ARG_out( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int));

  SAC_DISTMEM_BARRIER();
  SAC_PF_PRINT();
  SAC_CS_FINALIZE();
  SAC_MT_FINALIZE();
  SAC_CUDA_FINALIZE();
  SAC_HWLOC_FINALIZE();
  SAC_HM_PRINT();

  return( SAC_ND_READ( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0));
}
